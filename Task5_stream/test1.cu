#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"  
#include <iostream>
#include <stdio.h>  


const int N = 1 << 20;

__global__ void kernel(float *x, int n)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
        x[i] = sqrt(pow(3.14159,i));
    }
}

int main()
{
    const int num_streams = 8;
    hipStream_t streams[num_streams];
    float *data[num_streams];
    for (int i = 0; i < num_streams; i++) {
        hipStreamCreate(&streams[i]);
        hipMalloc(&data[i], N * sizeof(float));
        kernel<<<1, 64, 0, streams[i]>>>(data[i], N);
    }
    hipDeviceReset();
    hipEventSynchronize()
    return 0;
}